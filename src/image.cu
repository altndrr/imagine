#include "hip/hip_runtime.h"
#include "../include/common.h"
#include "../include/functions.cuh"
#include "../include/image.cuh"
#include "../include/kernel.h"
#include "../libs/stb/stb_image.h"
#include "../libs/stb/stb_image_write.h"
#include <hip/hip_runtime.h>
#include <queue>
#include <stdexcept>

Image::Image(const char *filename, bool grayscale) {
    _filename = filename;

    int desiredChannels = 3;
    if (grayscale) {
        desiredChannels = 1;
    }

    int w, h, c;
    unsigned char *data = stbi_load(_filename, &w, &h, &c, desiredChannels);

    if (!data) {
        return;
    }

    _width = w;
    _height = h;
    _channels = desiredChannels;

    _nBytes = w * h * desiredChannels * sizeof(unsigned char);

    // Allocate space for the host copy.
    _h_data = (unsigned char *)malloc(_nBytes);
    for (int i = 0; i < w * h * desiredChannels; i++) {
        _h_data[i] = data[i];
    }

    // Allocate space for the cuda copy.
    hipMalloc((unsigned char **)&_d_data, _nBytes);

    stbi_image_free(data);
}

Image::Image(const Image &obj) {
    _device = obj._device;
    _filename = obj._filename;
    _width = obj._width;
    _height = obj._height;
    _channels = obj._channels;
    _nBytes = _width * _height * _channels * sizeof(unsigned char);

    // Allocate space for the host copy.
    _h_data = (unsigned char *)malloc(_nBytes);
    for (int i = 0; i < _width * _height * _channels; i++) {
        _h_data[i] = obj._h_data[i];
    }

    // Allocate space for the cuda copy.
    hipMalloc((unsigned char **)&_d_data, _nBytes);
    hipMemcpy(_d_data, obj._d_data, _nBytes, hipMemcpyDeviceToDevice);
}

Image::~Image(void) {
    free(_h_data);
    hipFree(_d_data);
}

Image Image::operator-(const Image &obj) {
    // Return if images have different sizes.
    if (_width != obj._width or _height != obj._height or
        _channels != obj._channels) {
        throw std::invalid_argument("images have different sizes");
    }

    Image result(obj);
    result.setDevice(_device);

    if (strcmp(_device, _validDevices[0]) == 0) {
        differenceOnHost(result.getData(), getData(), getWidth(), getHeight(),
                         getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getSize() + threads.x - 1) / threads.x, 1);
        differenceOnDevice<<<blocks, threads>>>(result.getData(), getData(),
                                                getWidth(), getHeight(),
                                                getChannels());
    }

    return result;
}

int Image::getChannels() { return _channels; }

unsigned char *Image::getData() {
    if (strcmp(_device, _validDevices[0]) == 0) {
        return _h_data;
    } else {
        return _d_data;
    }
}

const char *Image::getDevice() { return _device; }

unsigned int *Image::getElement(int index) {
    unsigned int *values = new unsigned int[_channels];

    if (index >= _width * _height) {
        return NULL;
    }

    // Synchronize matrices if needed.
    // TODO optimise synchronization to avoid to check everytime when on cuda.
    if (strcmp(_device, _validDevices[1]) == 0 and isSynchronized() == 0) {
        hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
    }

    for (int c = 0; c < _channels; c++) {
        values[c] = _h_data[index * _channels + c];
    }

    return values;
}

unsigned int *Image::getElement(int row, int col) {
    if (row >= _height or col >= _width) {
        return NULL;
    }

    return getElement(row * _width + col);
}

const char *Image::getFilename() { return _filename; }

int Image::getHeight() { return _height; }

int Image::getSize() { return _width * _height * _channels; }

int Image::getWidth() { return _width; }

bool Image::isSynchronized() {
    unsigned char *h_d_data_copy = (unsigned char *)malloc(_nBytes);
    hipMemcpy(h_d_data_copy, _d_data, _nBytes, hipMemcpyDeviceToHost);

    float epsilon = 1.0E-8;
    int match = 1;
    for (int i = 0; i < getSize(); i++) {
        if (abs(_h_data[i] - h_d_data_copy[i]) > epsilon) {
            match = 0;
            break;
        }
    }

    free(h_d_data_copy);
    return match;
}

void Image::save(const char *filename) {
    // Synchronize matrices if needed.
    if (strcmp(_device, _validDevices[1]) == 0 and isSynchronized() == 0) {
        hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
    }

    stbi_write_png(filename, _width, _height, _channels, _h_data,
                   _width * _channels);
}

void Image::setDevice(const char *device) {
    if (arrayContains(_validDevices, device) == 0) {
        return;
    }

    if (strcmp(device, _device) != 0) {
        _device = device;

        if (strcmp(device, _validDevices[0]) == 0) {
            hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
        } else {
            hipMemcpy(_d_data, _h_data, _nBytes, hipMemcpyHostToDevice);
        }
    }
}

void Image::calcOpticalFlow(int *currentCorners, Image *previousFrame,
                            int *corners, int maxCorners, int levels) {
    Image gray(getFilename(), true);
    Image prevGray(previousFrame->getFilename(), true);
    gray.setDevice(getDevice());
    prevGray.setDevice(getDevice());

    if (strcmp(_device, _validDevices[0]) == 0) {
        unsigned char *currPyramidalScales[levels];
        unsigned char *prevPyramidalScales[levels];

        // Create the pyramidal scales.
        for (int l = 0; l < levels; l++) {
            int levelWidth = gray.getWidth() / pow(2, l);
            int levelHeight = gray.getHeight() / pow(2, l);
            currPyramidalScales[l] = new unsigned char[gray.getSize()];
            prevPyramidalScales[l] = new unsigned char[prevGray.getSize()];

            if (l == 0) {
                for (int i = 0; i < gray.getSize(); i++) {
                    currPyramidalScales[l][i] = gray.getData()[i];
                    prevPyramidalScales[l][i] = prevGray.getData()[i];
                }
            } else {
                scaleOnHost(currPyramidalScales[l], currPyramidalScales[l - 1],
                            0.5, levelWidth * 2, levelHeight * 2, 1);
                scaleOnHost(prevPyramidalScales[l], prevPyramidalScales[l - 1],
                            0.5, levelWidth * 2, levelHeight * 2, 1);
            }
        }

        opticalFLowOnHost(currentCorners, corners, maxCorners,
                          currPyramidalScales, prevPyramidalScales, levels,
                          gray.getWidth(), gray.getHeight());
    } else {
        // Copy corner arrays to device.
        size_t cornersBytes = maxCorners * sizeof(int);
        int *d_corners, *d_currCorners;
        hipMalloc((int **)&d_corners, cornersBytes);
        hipMalloc((int **)&d_currCorners, cornersBytes);
        hipMemcpy(d_corners, corners, cornersBytes, hipMemcpyHostToDevice);

        // Create the pyramidal scales.
        size_t pyramidBytes = gray.getSize() * sizeof(unsigned char);
        unsigned char *currPyramidalScales, *prevPyramidalScales;
        hipMalloc((unsigned char **)&currPyramidalScales,
                   levels * pyramidBytes);
        hipMalloc((unsigned char **)&prevPyramidalScales,
                   levels * pyramidBytes);
        int copyBlockSize = 1024;
        dim3 copyThreads(copyBlockSize, 1);
        dim3 copyBlocks(
            (gray.getWidth() * gray.getHeight() + copyThreads.x - 1) /
                copyThreads.x,
            1);

        for (int l = 0; l < levels; l++) {
            int levelWidth = gray.getWidth() / pow(2, l);
            int levelHeight = gray.getHeight() / pow(2, l);

            if (l == 0) {
                hipMemcpy(currPyramidalScales, gray.getData(), pyramidBytes,
                           hipMemcpyDeviceToDevice);
                hipMemcpy(prevPyramidalScales, prevGray.getData(),
                           pyramidBytes, hipMemcpyDeviceToDevice);
            } else {
                scaleOnDevice<<<copyBlocks, copyThreads>>>(
                    currPyramidalScales + l * gray.getSize(),
                    currPyramidalScales + (l - 1) * gray.getSize(), 0.5,
                    levelWidth * 2, levelHeight * 2, 1);
                scaleOnDevice<<<copyBlocks, copyThreads>>>(
                    prevPyramidalScales + l * gray.getSize(),
                    prevPyramidalScales + (l - 1) * gray.getSize(), 0.5,
                    levelWidth * 2, levelHeight * 2, 1);
            }
        }

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((maxCorners + copyThreads.x - 1) / copyThreads.x, 1);
        opticalFLowOnDevice<<<blocks, threads>>>(
            d_currCorners, d_corners, maxCorners, currPyramidalScales,
            prevPyramidalScales, levels, gray.getSize(), gray.getWidth(),
            gray.getHeight());

        hipMemcpy(currentCorners, d_currCorners, cornersBytes,
                   hipMemcpyDeviceToHost);

        // Free memory.
        hipFree(d_corners);
        hipFree(d_currCorners);
        hipFree(currPyramidalScales);
        hipFree(prevPyramidalScales);
    }
}

void Image::convolution(float *kernel, int kernelSide) {
    unsigned char *dataCopy;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Create a copy of the data on host.
        dataCopy = (unsigned char *)malloc(_nBytes);
        for (int i = 0; i < getSize(); i++) {
            dataCopy[i] = getData()[i];
        }

        convolutionOnHost(getData(), dataCopy, kernel, kernelSide, getWidth(),
                          getHeight(), getChannels());
    } else {
        // Create a copy of the data on device.
        hipMalloc((unsigned char **)&dataCopy, _nBytes);
        hipMemcpy(dataCopy, getData(), _nBytes, hipMemcpyDeviceToDevice);

        // Copy kernel to device.
        float *d_kernel;
        hipMalloc((float **)&d_kernel,
                   kernelSide * kernelSide * sizeof(float));
        hipMemcpy(d_kernel, kernel, kernelSide * kernelSide * sizeof(float),
                   hipMemcpyHostToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        convolutionOnDevice<<<blocks, threads>>>(getData(), dataCopy, d_kernel,
                                                 kernelSide, getWidth(),
                                                 getHeight(), getChannels());

        // Free memory.
        hipFree(dataCopy);
    }
}

void Image::drawLine(int index1, int index2, int radius, int *color,
                     int colorSize) {
    if (index1 < 0 or index2 < 0) {
        return;
    }

    int x1 = (int)(index1 / getWidth());
    int y1 = (index1 % getWidth());
    int x2 = (int)(index2 / getWidth());
    int y2 = (index2 % getWidth());
    this->drawLine(x1, y1, x2, y2, radius, color, colorSize);
}

void Image::drawLine(int x1, int y1, int x2, int y2, int radius, int *color,
                     int colorSize) {
    if (strcmp(_device, _validDevices[0]) == 0) {
        drawLineOnHost(getData(), x1, y1, x2, y2, radius, color, colorSize,
                       getWidth(), getHeight(), getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);

        size_t colorBytes = colorSize * sizeof(int);
        int *d_color;
        hipMalloc((int **)&d_color, colorBytes);
        hipMemcpy(d_color, color, colorBytes, hipMemcpyHostToDevice);

        drawLineOnDevice<<<blocks, threads>>>(getData(), x1, y1, x2, y2, radius,
                                              d_color, colorSize, getWidth(),
                                              getHeight(), getChannels());
    }
}

void Image::drawPoint(int index, int radius, int *color, int colorSize) {
    if (index < 0) {
        return;
    }

    int x = (int)(index / getWidth());
    int y = (index % getWidth());
    this->drawPoint(x, y, radius, color, colorSize);
}

void Image::drawPoint(int x, int y, int radius, int *color, int colorSize) {
    if (strcmp(_device, _validDevices[0]) == 0) {
        drawPointOnHost(getData(), x, y, radius, color, colorSize, getWidth(),
                        getHeight(), getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);

        size_t colorBytes = colorSize * sizeof(int);
        int *d_color;
        hipMalloc((int **)&d_color, colorBytes);
        hipMemcpy(d_color, color, colorBytes, hipMemcpyHostToDevice);

        drawPointOnDevice<<<blocks, threads>>>(getData(), x, y, radius, d_color,
                                               colorSize, getWidth(),
                                               getHeight(), getChannels());
    }
}

void Image::findHomography(float *A, int *currentCorners, int *previousCorners,
                           int maxCorners) {
    const int maxIter = 2000;
    const int N_POINTS = 3;
    const int SPACE_DIM = 2;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Estimate maxIter different rigid transformations.
        // The algorithm estimates a matrix using a triplet of points.
        float *matrices = new float[N_POINTS * (SPACE_DIM + 1) * maxIter];
        float *scores = new float[maxIter];
        findHomographyRANSACOnHost(matrices, scores, maxIter, currentCorners,
                                   previousCorners, maxCorners, getWidth(),
                                   getHeight());

        int bestMatrix = -1;
        float minError = INFINITY;

        for (int i = 0; i < maxIter; i++) {
            if (scores[i] < minError) {
                bestMatrix = i;
                minError = scores[i];
            }
        }

        int offset = bestMatrix * (N_POINTS * (SPACE_DIM + 1));
        for (int i = 0; i < N_POINTS * (SPACE_DIM + 1); i++) {
            if (minError < INFINITY) {
                A[i] = matrices[offset + i];
            } else {
                // If the minError is INFINITY, then set the transformation to
                // the identity matrix to avoid any type of transformation.
                int side = sqrt(N_POINTS * (SPACE_DIM + 1));
                A[i] = int(i % side == (int)i / side);
            }
        }

        delete[] matrices;
        delete[] scores;
    }
}

void Image::goodFeaturesToTrack(int *corners, int maxCorners,
                                float qualityLevel, float minDistance) {
    Image gradX(getFilename(), true);
    Image gradY(getFilename(), true);
    gradX.setDevice(getDevice());
    gradY.setDevice(getDevice());

    int side;
    float *sobelX, *sobelY;
    Kernel::SobelX(&sobelX, &side);
    Kernel::SobelY(&sobelY);
    gradX.convolution(sobelX, side);
    gradY.convolution(sobelY, side);

    int scoreSize = getWidth() * getHeight();
    float *scoreMatrix = new float[scoreSize];

    if (strcmp(_device, _validDevices[0]) == 0) {
        cornerScoreOnHost(gradX.getData(), gradY.getData(), scoreMatrix,
                          getWidth(), getHeight());
    } else {
        // Copy corner array to device.
        size_t scoreMatrixBytes = scoreSize * sizeof(float);
        float *d_scoreMatrix;
        hipMalloc((float **)&d_scoreMatrix, scoreMatrixBytes);
        hipMemcpy(d_scoreMatrix, scoreMatrix, scoreMatrixBytes,
                   hipMemcpyHostToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((scoreSize + threads.x - 1) / threads.x, 1);
        cornerScoreOnDevice<<<blocks, threads>>>(gradX.getData(),
                                                 gradY.getData(), d_scoreMatrix,
                                                 getWidth(), getHeight());

        // Copy result to host.
        hipMemcpy(scoreMatrix, d_scoreMatrix, scoreMatrixBytes,
                   hipMemcpyDeviceToHost);
        hipFree(d_scoreMatrix);
    }

    // Create a priority queue of the scores and store the highest score.
    std::priority_queue<std::pair<float, int>> qR;
    float strongestScore = 0.00;
    for (int i = 0; i < scoreSize; i++) {
        // Skip nan values.
        if (scoreMatrix[i] != scoreMatrix[i]) {
            continue;
        }

        qR.push(std::pair<float, int>(scoreMatrix[i], i));
        if (strongestScore < scoreMatrix[i]) {
            strongestScore = scoreMatrix[i];
        }
    }

    // Extract the top-K corners.
    float threshold = strongestScore * qualityLevel;
    for (int i = 0; i < maxCorners; ++i) {
        corners[i] = -1;
        float kValue;
        int kIndex;
        bool isDistant;

        do {
            kValue = qR.top().first;
            kIndex = qR.top().second;
            isDistant = true;

            // Evaluate the Euclidean distance to the previous corners.
            int j = 0;
            while (j < i and isDistant) {
                int otherIndex = corners[j];
                int dx =
                    ((int)otherIndex / getWidth()) - ((int)kIndex / getWidth());
                int dy = (otherIndex % getWidth()) - (kIndex % getWidth());
                int dist = sqrt(pow(dx, 2) + pow(dy, 2));

                isDistant = dist > minDistance;
                j++;
            }

            if (isDistant) {
                // Add only if score is high enough.
                if (kValue >= threshold) {
                    corners[i] = kIndex;
                }
            }

            qR.pop();
        } while (not isDistant);
    }
}

unsigned char *Image::histogram() {
    size_t histBytes = PIXEL_VALUES * getChannels() * sizeof(unsigned char);
    unsigned char *histogram = (unsigned char *)malloc(histBytes);

    for (int i = 0; i < PIXEL_VALUES * getChannels(); i++) {
        histogram[i] = 0;
    }

    if (strcmp(_device, _validDevices[0]) == 0) {
        histogramOnHost(histogram, getData(), getWidth(), getHeight(),
                        getChannels());
    } else {
        // Copy histogram to device.
        unsigned char *d_histogram;
        hipMalloc((unsigned char **)&d_histogram, histBytes);
        hipMemcpy(d_histogram, histogram, histBytes, hipMemcpyHostToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getSize() + threads.x - 1) / threads.x, 1);
        histogramOnDevice<<<blocks, threads>>>(
            d_histogram, getData(), getWidth(), getHeight(), getChannels());

        // Copy result to host.
        hipMemcpy(histogram, d_histogram, histBytes, hipMemcpyDeviceToHost);
        hipFree(d_histogram);
    }

    return histogram;
}

void Image::rotate(double degree) {
    double rad = degree * (M_PI / 180);
    unsigned char *dataCopy;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Create a copy of the data on host.
        dataCopy = (unsigned char *)malloc(_nBytes);
        for (int i = 0; i < getSize(); i++) {
            dataCopy[i] = getData()[i];
        }

        rotateOnHost(getData(), dataCopy, rad, getWidth(), getHeight(),
                     getChannels());
    } else {
        // Copy histogram to device.
        hipMalloc((unsigned char **)&dataCopy, _nBytes);
        hipMemcpy(dataCopy, _d_data, _nBytes, hipMemcpyDeviceToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        rotateOnDevice<<<blocks, threads>>>(
            getData(), dataCopy, rad, getWidth(), getHeight(), getChannels());
    }
}

void Image::scale(float ratio) {
    // Return if ratio is invalid.
    if (ratio == 1.0 or ratio < 0.0) {
        return;
    }

    unsigned char *newData;
    int newWidth = int(getWidth() * ratio);
    int newHeight = int(getHeight() * ratio);
    int newBytes = newWidth * newHeight * getChannels() * sizeof(unsigned char);

    if (strcmp(_device, _validDevices[0]) == 0) {
        newData = (unsigned char *)malloc(newBytes);
        scaleOnHost(newData, getData(), ratio, getWidth(), getHeight(),
                    getChannels());

        // Update data both on device and on host.
        free(_h_data);
        _h_data = newData;
        hipFree(_d_data);
        hipMalloc((unsigned char **)&_d_data, newBytes);
        hipMemcpy(_d_data, _h_data, newBytes, hipMemcpyHostToDevice);
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((newWidth * newHeight + threads.x - 1) / threads.x, 1);

        hipMalloc((unsigned char **)&newData, newBytes);
        scaleOnDevice<<<blocks, threads>>>(
            newData, getData(), ratio, getWidth(), getHeight(), getChannels());

        // Update data both on device and on host.
        hipFree(_d_data);
        hipMalloc((unsigned char **)&_d_data, newBytes);
        hipMemcpy(_d_data, newData, newBytes, hipMemcpyDeviceToDevice);
        free(_h_data);
        _h_data = (unsigned char *)malloc(newBytes);
        hipMemcpy(_h_data, _d_data, newBytes, hipMemcpyDeviceToHost);
    }

    // Update other attributes.
    _width = newWidth;
    _height = newHeight;
    _nBytes = newBytes;
}

void Image::translate(int px, int py) {
    unsigned char *dataCopy;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Create a copy of the data on host.
        dataCopy = (unsigned char *)malloc(_nBytes);
        for (int i = 0; i < getSize(); i++) {
            dataCopy[i] = getData()[i];
        }

        translateOnHost(getData(), dataCopy, px, py, getWidth(), getHeight(),
                        getChannels());
    } else {
        // Copy histogram to device.
        hipMalloc((unsigned char **)&dataCopy, _nBytes);
        hipMemcpy(dataCopy, _d_data, _nBytes, hipMemcpyDeviceToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        translateOnDevice<<<blocks, threads>>>(getData(), dataCopy, px, py,
                                               getWidth(), getHeight(),
                                               getChannels());
    }
}

void Image::transpose() {
    // Return if width and height are different.
    if (getWidth() != getHeight()) {
        throw std::invalid_argument("width and height must have the same size");
    }

    if (strcmp(_device, _validDevices[0]) == 0) {
        transposeOnHost(getData(), getWidth(), getHeight(), getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        transposeOnDevice<<<blocks, threads>>>(getData(), getWidth(),
                                               getHeight(), getChannels());
    }
}
