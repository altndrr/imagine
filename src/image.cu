#include "hip/hip_runtime.h"
#include "../include/image.cuh"
#include "../include/common.h"
#include "../include/functions.cuh"
#include "../libs/stb/stb_image.h"
#include "../libs/stb/stb_image_write.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

Image::Image(const char *filename) {
    _filename = filename;

    int w, h, c;
    unsigned char *data = stbi_load(_filename, &w, &h, &c, 3);

    if (!data) {
        return;
    }

    _width = w;
    _height = h;
    _channels = c;

    _nBytes = w * h * c * sizeof(unsigned char);

    // Allocate space for the host copy.
    _h_data = (unsigned char *) malloc(_nBytes);
    for (int i = 0; i < w * h * c; i++) {
        _h_data[i] = data[i];
    }

    // Allocate space for the cuda copy.
    hipMalloc((unsigned char **) &_d_data, _nBytes);

    stbi_image_free(data);
}

Image::Image(const Image &obj) {
    _device = obj._device;
    _filename = obj._filename;
    _width = obj._width;
    _height = obj._height;
    _channels = obj._channels;
    _nBytes = _width * _height * _channels * sizeof(unsigned char);

    // Allocate space for the host copy.
    _h_data = (unsigned char *) malloc(_nBytes);
    for (int i = 0; i < _width * _height * _channels; i++) {
        _h_data[i] = obj._h_data[i];
    }

    // Allocate space for the cuda copy.
    hipMalloc((unsigned char **) &_d_data, _nBytes);
    hipMemcpy(_d_data, obj._d_data, _nBytes, hipMemcpyDeviceToDevice);
}

Image::~Image(void) {
    free(_h_data);
    hipFree(_d_data);
}

Image Image::operator-(const Image &obj) {
    // Return if images have different sizes.
    if (_width != obj._width or _height != obj._height or _channels != obj._channels) {
        throw std::invalid_argument("images have different sizes");
    }

    Image result(obj);
    result.setDevice(_device);

    if (strcmp(_device, _validDevices[0]) == 0) {
        differenceOnHost(result.getData(), getData(), getWidth(), getHeight(), getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getSize() + threads.x - 1) / threads.x, 1);
        differenceOnDevice<<<blocks, threads>>>(result.getData(), getData(), getWidth(), getHeight(), getChannels());
    }

    return result;
}

int Image::getChannels() {
    return _channels;
}

unsigned char *Image::getData() {
    if (strcmp(_device, _validDevices[0]) == 0) {
        return _h_data;
    } else {
        return _d_data;
    }
}

const char *Image::getDevice() {
    return _device;
}

unsigned int *Image::getElement(int index) {
    unsigned int *values = new unsigned int[_channels];

    if (index >= _width * _height) {
        return NULL;
    }

    // Synchronize matrices if needed.
    // TODO optimise synchronization to avoid to check everytime when on cuda.
    if (strcmp(_device, _validDevices[1]) == 0 and isSynchronized() == 0) {
        hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
    }

    for (int c = 0; c < _channels; c++) {
        values[c] = _h_data[index * _channels + c];
    }

    return values;
}

unsigned int *Image::getElement(int row, int col) {
    if (row >= _height or col >= _width) {
        return NULL;
    }

    return getElement(row * _width + col);
}

const char *Image::getFilename() {
    return _filename;
}

int Image::getHeight() {
    return _height;
}

int Image::getSize() {
    return _width * _height * _channels;
}

int Image::getWidth() {
    return _width;
}

bool Image::isSynchronized() {
    unsigned char *h_d_data_copy = (unsigned char *) malloc(_nBytes);
    hipMemcpy(h_d_data_copy, _d_data, _nBytes, hipMemcpyDeviceToHost);

    float epsilon = 1.0E-8;
    int match = 1;
    for (int i = 0; i < getSize(); i++) {
        if (abs(_h_data[i] - h_d_data_copy[i]) > epsilon) {
            match = 0;
            break;
        }
    }

    free(h_d_data_copy);
    return match;
}

void Image::save(const char *filename) {
    // Synchronize matrices if needed.
    if (strcmp(_device, _validDevices[1]) == 0 and isSynchronized() == 0) {
        hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
    }

    stbi_write_png(filename, _width, _height, _channels, _h_data, _width * _channels);
}

void Image::setDevice(const char *device) {
    if (arrayContains(_validDevices, device) == 0) {
        return;
    }

    if (strcmp(device, _device) != 0) {
        _device = device;

        if (strcmp(device, _validDevices[0]) == 0) {
            hipMemcpy(_h_data, _d_data, _nBytes, hipMemcpyDeviceToHost);
        } else {
            hipMemcpy(_d_data, _h_data, _nBytes, hipMemcpyHostToDevice);
        }
    }
}

void Image::convolution(float *kernel, int kernelSide) {
    unsigned char *dataCopy;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Create a copy of the data on host.
        dataCopy = (unsigned char *) malloc(_nBytes);
        for (int i = 0; i < getSize(); i++) {
            dataCopy[i] = getData()[i];
        }

        convolutionOnHost(getData(), dataCopy, kernel, kernelSide, getWidth(), getHeight(),
                          getChannels());
    } else {
        // Create a copy of the data on device.
        hipMalloc((unsigned char **) &dataCopy, _nBytes);
        hipMemcpy(dataCopy, getData(), _nBytes, hipMemcpyDeviceToDevice);

        // Copy kernel to device.
        float *d_kernel;
        hipMalloc((float **) &d_kernel, kernelSide * kernelSide * sizeof(float));
        hipMemcpy(d_kernel, kernel, kernelSide * kernelSide * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        convolutionOnDevice<<<blocks, threads>>>(getData(), dataCopy, d_kernel, kernelSide, getWidth(), getHeight(),
                                                 getChannels());

        // Free memory.
        hipFree(dataCopy);
    }
}

unsigned char *Image::histogram() {
    size_t histBytes = PIXEL_VALUES * getChannels() * sizeof(unsigned char);
    unsigned char *histogram = (unsigned char *) malloc(histBytes);

    for (int i = 0; i < PIXEL_VALUES * getChannels(); i++) {
        histogram[i] = 0;
    }

    if (strcmp(_device, _validDevices[0]) == 0) {
        histogramOnHost(histogram, getData(), getWidth(), getHeight(), getChannels());
    } else {
        // Copy histogram to device.
        unsigned char *d_histogram;
        hipMalloc((unsigned char **) &d_histogram, histBytes);
        hipMemcpy(d_histogram, histogram, histBytes, hipMemcpyHostToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getSize() + threads.x - 1) / threads.x, 1);
        histogramOnDevice<<<blocks, threads>>>(d_histogram, getData(), getWidth(), getHeight(), getChannels());

        // Copy result to host.
        hipMemcpy(histogram, d_histogram, histBytes, hipMemcpyDeviceToHost);
        hipFree(d_histogram);
    }

    return histogram;
}

void Image::rotate(double degree) {
    double rad = degree * (M_PI / 180);
    unsigned char *dataCopy;

    if (strcmp(_device, _validDevices[0]) == 0) {
        // Create a copy of the data on host.
        dataCopy = (unsigned char *) malloc(_nBytes);
        for (int i = 0; i < getSize(); i++) {
            dataCopy[i] = getData()[i];
        }

        rotateOnHost(getData(), dataCopy, rad, getWidth(), getHeight(), getChannels());
    } else {
        // Copy histogram to device.
        hipMalloc((unsigned char **) &dataCopy, _nBytes);
        hipMemcpy(dataCopy, _d_data, _nBytes, hipMemcpyDeviceToDevice);

        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        rotateOnDevice<<<blocks, threads>>>(getData(), dataCopy, rad, getWidth(), getHeight(), getChannels());
    }
}

void Image::transpose() {
    // Return if width and height are different.
    if (getWidth() != getHeight()) {
        throw std::invalid_argument("width and height must have the same size");
    }

    if (strcmp(_device, _validDevices[0]) == 0) {
        transposeOnHost(getData(), getWidth(), getHeight(), getChannels());
    } else {
        int blockSize = 1024;
        dim3 threads(blockSize, 1);
        dim3 blocks((getWidth() * getHeight() + threads.x - 1) / threads.x, 1);
        transposeOnDevice<<<blocks, threads>>>(getData(), getWidth(), getHeight(), getChannels());
    }
}
